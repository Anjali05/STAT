#include "hip/hip_runtime.h"
/*
Copyright (c) 2007-2018, Lawrence Livermore National Security, LLC.
Produced at the Lawrence Livermore National Laboratory
Written by Gregory Lee [lee218@llnl.gov], Dorian Arnold, Matthew LeGendre, Dong Ahn, Bronis de Supinski, Barton Miller, Martin Schulz, Niklas Nielson, Nicklas Bo Jensen, Jesper Nielson, and Sven Karlsson.
LLNL-CODE-750488.
All rights reserved.

This file is part of STAT. For details, see http://www.github.com/LLNL/STAT. Please also read STAT/LICENSE.

Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:

        Redistributions of source code must retain the above copyright notice, this list of conditions and the disclaimer below.
        Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the disclaimer (as noted below) in the documentation and/or other materials provided with the distribution.
        Neither the name of the LLNS/LLNL nor the names of its contributors may be used to endorse or promote products derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL LAWRENCE LIVERMORE NATIONAL SECURITY, LLC, THE U.S. DEPARTMENT OF ENERGY OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <assert.h>

#ifdef USEMPI
#include "mpi.h"

char hostname[256];
int sleeptime = -1;
void do_SendOrStall(int to, int tag, int rank, int* buf, MPI_Request* req, int n)
{
    int i;
    if (rank == 1)
    {
        if (sleeptime == -1)
        {
            printf("%s, MPI task %d of %d stalling\n", hostname, rank, n);
            fflush(stdout);
            while(1) ;
        }
        else
        {
            for (i = sleeptime/10; i > 0; i = i - 1)
            {
                printf("%s, MPI task %d of %d stalling for %d of %d seconds\n", hostname, rank, n, i*10, sleeptime);
                fflush(stdout);
                sleep(10);
            }
            printf("%s, MPI task %d of %d proceeding\n", hostname, rank, n);
            fflush(stdout);
        }
    }

    MPI_Isend(buf, 1, MPI_INT, to, tag, MPI_COMM_WORLD, req);
}

void do_Receive(int from, int tag, int* buf, MPI_Request* req)
{
    MPI_Irecv(buf, 1, MPI_INT, from, tag, MPI_COMM_WORLD, req);
}
#endif

#define N (32*10)
#define THREADS_PER_BLOCK 32

__device__ void foo()
{
   int i, x, y;
#ifdef CRASH
   assert(0);
#endif
#ifdef NOHANG
   for (i = 0; i <= 1000000; i++)
#else
   for (i = 0; i >= 0; i++)
#endif
   {
    x = i;
    y = x + 1;
   }
}

__device__ void bar()
{
    foo();
}

__global__
void add(int *a, int *b)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x, *z;
    if (i<N)
    {
        b[i] = 2*a[i];
#ifdef CRASH
        z[i] = b[i];
        b[i] = z[i];
#endif
    }
    if (threadIdx.x % 32 == 0)
        bar();
    else
        bar();
#ifdef CRASH
    free(z);
    free(z);
    free(a);
    free(b);
#endif
}

__global__
void add2(int *a, int *b)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<N)
        b[i] = 2*a[i];
    bar();
}

int main(int argc, char **argv)
{
    int ha[N], hb[N], hc[N];
    gethostname(hostname, 256);
#ifdef USEMPI
    int next, prev, buf[2], tag=2;
    MPI_Request reqs[2];
    MPI_Status stats[2];
    int numtasks, rank;
    MPI_Init(&argc,&argv);
    MPI_Comm_size(MPI_COMM_WORLD, &numtasks);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    printf("Hello world from %s %d/%d\n", hostname, rank, numtasks);
#else
    printf("Hello serial world from %s\n", hostname);
#endif
    fflush(stdout);
    int *da, *db;
    hipMalloc((void **)&da, N*sizeof(int));
    hipMalloc((void **)&db, N*sizeof(int));
    int *dc, *dd;
    hipMalloc((void **)&dc, N*sizeof(int));
    hipMalloc((void **)&dd, N*sizeof(int));
    for (int i = 0; i<N; ++i)
        ha[i] = i;
    hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dc, ha, N*sizeof(int), hipMemcpyHostToDevice);
#ifdef USEMPI
    if (rank % 2 == 0)
#endif
#ifdef HOSTCRASH
    {
        free(da);
        free(da);
        free(0);
    }
#endif
    add<<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(da, db);
    add2<<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(dc, dd);

#ifdef USEMPI

    if (argc > 1)
        sleeptime = atoi(argv[1]);
    prev = rank-1;
    next = rank+1;
    if (rank == 0)
        prev = numtasks - 1;
    if (rank == (numtasks - 1))
        next = 0;

    do_Receive(prev, tag, &buf[0], &reqs[0]);
    do_SendOrStall(next, tag, rank, &buf[1], &reqs[1], numtasks);
    MPI_Waitall(2, reqs, stats);

    MPI_Barrier(MPI_COMM_WORLD);
#endif
    hipMemcpy(hc, dd, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i<N; ++i)
      if (i == 99)
        printf("%d\n", hb[i], hc[i]);
    fflush(stdout);
    hipFree(da);
    hipFree(db);
#ifdef USEMPI
    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Finalize();
#endif
    return 0;
}
