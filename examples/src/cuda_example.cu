
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#ifdef USEMPI
#include "mpi.h"
#endif

#define N (32*10)
#define THREADS_PER_BLOCK 32

__device__ void foo()
{
   int i, x, y;
#ifdef NOHANG
   for (i = 0; i <= 1000000; i++)
#else
   for (i = 0; i >=0; i++)
#endif
   {
    x = i;
    y = x + 1;
   }
}

__device__ void bar()
{
    foo();
}

__global__
void add(int *a, int *b) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<N) {
        b[i] = 2*a[i];
    }
    if (threadIdx.x % 32 == 0)
        bar();
    else
        bar();
}

__global__
void add2(int *a, int *b) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<N) {
        b[i] = 2*a[i];
    }
    bar();
}

int main(int argc, char **argv) {
    int ha[N], hb[N], hc[N];
#ifdef USEMPI
    int numtasks, rank;
    char hostname[256];
    MPI_Init(&argc,&argv);
    MPI_Comm_size(MPI_COMM_WORLD, &numtasks);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    gethostname(hostname, 256);
    printf("Hello world from %s %d/%d\n", hostname, rank, numtasks);
#endif
    int *da, *db;
    hipMalloc((void **)&da, N*sizeof(int));
    hipMalloc((void **)&db, N*sizeof(int));
    int *dc, *dd;
    hipMalloc((void **)&dc, N*sizeof(int));
    hipMalloc((void **)&dd, N*sizeof(int));
    for (int i = 0; i<N; ++i)
        ha[i] = i;
    hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dc, ha, N*sizeof(int), hipMemcpyHostToDevice);
#ifdef USEMPI
    if (rank % 2 == 0)
#endif
    add<<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(da, db);
    add2<<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(dc, dd);
    hipMemcpy(hc, dd, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i<N; ++i) {
      if (i == 99)
        printf("%d\n", hb[i], hc[i]);
    }
    hipFree(da);
    hipFree(db);
    return 0;
}
