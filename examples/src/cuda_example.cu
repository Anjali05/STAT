/*
Copyright (c) 2007-2017, Lawrence Livermore National Security, LLC.
Produced at the Lawrence Livermore National Laboratory
Written by Gregory Lee [lee218@llnl.gov], Dorian Arnold, Matthew LeGendre, Dong Ahn, Bronis de Supinski, Barton Miller, Martin Schulz, Niklas Nielson, Nicklas Bo Jensen, Jesper Nielson, and Sven Karlsson.
LLNL-CODE-727016.
All rights reserved.

This file is part of STAT. For details, see http://www.github.com/LLNL/STAT. Please also read STAT/LICENSE.

Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:

        Redistributions of source code must retain the above copyright notice, this list of conditions and the disclaimer below.
        Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the disclaimer (as noted below) in the documentation and/or other materials provided with the distribution.
        Neither the name of the LLNS/LLNL nor the names of its contributors may be used to endorse or promote products derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL LAWRENCE LIVERMORE NATIONAL SECURITY, LLC, THE U.S. DEPARTMENT OF ENERGY OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#ifdef USEMPI
#include "mpi.h"
#endif

#define N (32*10)
#define THREADS_PER_BLOCK 32

__device__ void foo()
{
   int i, x, y;
#ifdef NOHANG
   for (i = 0; i <= 1000000; i++)
#else
   for (i = 0; i >=0; i++)
#endif
   {
    x = i;
    y = x + 1;
   }
}

__device__ void bar()
{
    foo();
}

__global__
void add(int *a, int *b)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<N)
        b[i] = 2*a[i];
    if (threadIdx.x % 32 == 0)
        bar();
    else
        bar();
}

__global__
void add2(int *a, int *b)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i<N)
        b[i] = 2*a[i];
    bar();
}

int main(int argc, char **argv)
{
    int ha[N], hb[N], hc[N];
#ifdef USEMPI
    int numtasks, rank;
    char hostname[256];
    MPI_Init(&argc,&argv);
    MPI_Comm_size(MPI_COMM_WORLD, &numtasks);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    gethostname(hostname, 256);
    printf("Hello world from %s %d/%d\n", hostname, rank, numtasks);
#endif
    int *da, *db;
    hipMalloc((void **)&da, N*sizeof(int));
    hipMalloc((void **)&db, N*sizeof(int));
    int *dc, *dd;
    hipMalloc((void **)&dc, N*sizeof(int));
    hipMalloc((void **)&dd, N*sizeof(int));
    for (int i = 0; i<N; ++i)
        ha[i] = i;
    hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dc, ha, N*sizeof(int), hipMemcpyHostToDevice);
#ifdef USEMPI
    if (rank % 2 == 0)
#endif
    add<<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(da, db);
    add2<<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(dc, dd);
    hipMemcpy(hc, dd, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i<N; ++i)
      if (i == 99)
        printf("%d\n", hb[i], hc[i]);
    hipFree(da);
    hipFree(db);
    return 0;
}
